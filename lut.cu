#include "hip/hip_runtime.h"
#include "lut.h"

float3 Interpolate(float3 &v0, float3 &v1, float f)
{
	float3 out = make_float3(
		InterpolateHelper(v0.x, v1.x, f),
		InterpolateHelper(v0.y, v1.y, f),
		InterpolateHelper(v0.z, v1.z, f)
	);

	return out;
}

// interpolation: v0 + (v1 - v0) * f, v0/v1 = RGB component, f = weight
float InterpolateHelper(float v0, float v1, float f)
{
	return v0 + (v1 - v0) * f;
}

float3 trilinearInterpolation(float3 pos, float* lut, const size_t lutSize, uint8_t bitDepth) {
    
    const size_t totalLutSize = lutSize * lutSize * lutSize * 3;

    float R = static_cast<float>(pos.x);
    float G = static_cast<float>(pos.y);
    float B = static_cast<float>(pos.z);
    
    float normFactor = static_cast<float>((1 << bitDepth) - 1);

	R /= normFactor;
	G /= normFactor;
	B /= normFactor;
    
    // convert from point to grid coordinates
	float x = R * (lutSize - 1);
	float y = G * (lutSize - 1);
	float z = B * (lutSize - 1);

	// round down for coordinates
	int intX = static_cast<int>(x);
	int intY = static_cast<int>(y);
	int intZ = static_cast<int>(z);

    // difference between x and intX (floor of x)
    delta d;
    d.x = x - intX;
    d.y = y - intY;
    d.z = z - intZ;
    
    // calculate vertices in cube
	// cxyz is each coordinate, where xyz is a binary representation of a number
	// x changes fastest and represents the offset into a row, so add 1 each time it is set
	// y changes second fastest and represents a row in the cube. Add the lut size (the size of one dimension) when it is set
	// z changes the slowest and represents the plane in the cube. Add the square of the lut size when it is set

	size_t xOffset = 1 * 3;
	size_t yOffset = lutSize * 3;
	size_t zOffset = lutSize * lutSize * 3;

    // To prevent the out of bounds, we will start the index from 0 + difference again whenever an index is out of bounds
    
    // multiply by the number of components in an RGB triplet (3)
	// c000
	size_t index0 = (intZ * zOffset + intY * yOffset + intX * xOffset);

	// c001
	size_t index1 = (index0 + zOffset);
    if (index1 >= totalLutSize) {
        index1 -= totalLutSize;
    }

	// c010
	size_t index2 = (index0 + yOffset);
    if (index2 >= totalLutSize) {
        index2 -= totalLutSize;
    }

	// c011
	size_t index3 = (index2 + zOffset);
    if (index3 >= totalLutSize) {
        index3 -= totalLutSize;
    }

	// c100
	size_t index4 = (index0 + xOffset);
    if (index4 >= totalLutSize) {
        index4 -= totalLutSize;
    }

	// c101
	size_t index5 = (index4 + zOffset);
    if (index5 >= totalLutSize) {
        index5 -= totalLutSize;
    }

	// c110
	size_t index6 = (index4 + yOffset);
    if (index6 >= totalLutSize) {
        index6 -= totalLutSize;
    }

	// c111
	size_t index7 = (index6 + zOffset);
    if (index7 >= totalLutSize) {
        index7 -= totalLutSize;
    }


    float3 c000 = make_float3(static_cast<float>(lut[index0]), static_cast<float>(lut[index0 + 1]), static_cast<float>(lut[index0 + 2]));
	float3 c001 = make_float3(static_cast<float>(lut[index1]), static_cast<float>(lut[index1 + 1]), static_cast<float>(lut[index1 + 2]));
	float3 c010 = make_float3(static_cast<float>(lut[index2]), static_cast<float>(lut[index2 + 1]), static_cast<float>(lut[index2 + 2]));
	float3 c011 = make_float3(static_cast<float>(lut[index3]), static_cast<float>(lut[index3 + 1]), static_cast<float>(lut[index3 + 2]));
	float3 c100 = make_float3(static_cast<float>(lut[index4]), static_cast<float>(lut[index4 + 1]), static_cast<float>(lut[index4 + 2]));
	float3 c101 = make_float3(static_cast<float>(lut[index5]), static_cast<float>(lut[index5 + 1]), static_cast<float>(lut[index5 + 2]));
	float3 c110 = make_float3(static_cast<float>(lut[index6]), static_cast<float>(lut[index6 + 1]), static_cast<float>(lut[index6 + 2]));
	float3 c111 = make_float3(static_cast<float>(lut[index7]), static_cast<float>(lut[index7 + 1]), static_cast<float>(lut[index7 + 2]));

    // c00 -> interpolate c000 and c100
	float3 c00 = Interpolate(c000, c100, d.x);

	// c01 -> interpolate c001 and c101
	float3 c01 = Interpolate(c001, c101, d.x);

	// c10 -> interpolate c010 and c110
	float3 c10 = Interpolate(c010, c110, d.x);

	// c11 -> interpolate c011 and c111
	float3 c11 = Interpolate(c011, c111, d.x);

	// c0 -> interpolate c00 and c10
	float3 c0 = Interpolate(c00, c10, d.y);

	// c1 -> interpolate c01 and c11
	float3 c1 = Interpolate(c01, c11, d.y);

	// c -> interpolate c0 and c1
	float3 c = Interpolate(c0, c1, d.z);

    // create a vector out of all the coefficients
    float3 result = make_float3(static_cast<uint8_t>(c.x * normFactor), static_cast<uint8_t>(c.y * normFactor), static_cast<uint8_t>(c.z * normFactor));
    return result;
}

// Load the .cube LUT file
void getLutValues(std::string filename, int lutSize, float* values) {
    std::ifstream file(filename);
    std::cout << "Reading LUT file..." << std::endl;
    if (!file.is_open()) {
        std::cerr << "Error opening the file." << std::endl;
        exit(1);
    }
    
    // Read and ignore the title line
    std::string title;
    std::getline(file, title);
    title = title.substr(0, title.size() - 1); // Remove trailing newline

    // Read the LUT size
    std::string lutSizeLine;
    std::getline(file, lutSizeLine);
    std::istringstream lutSizeStream(lutSizeLine);
    std::string discard;
    int lut_size;

    lutSizeStream >> discard >> lut_size;
    
    if (lut_size != lutSize) {
        std::cerr << "Error: LUT size does not match the specified size." << std::endl;
        exit(1);
    }

    // Count the number of elements
    int elementCount = 0;
    std::string line;

    while (std::getline(file, line)) {
        std::istringstream valuesStream(line);
        std::string value;

        while (valuesStream >> value) {
            values[elementCount] = std::stof(value);
            elementCount++;
        }
    }

    file.close();
}

// CUDA kernel to apply LUT to each pixel in parallel
__global__
void applyLUTKernel(const uint8_t* input, uint8_t* output, int frameSize, const uint8_t* lut) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    frameSize >>= 1;
    for(int i = index; i < frameSize; i += stride) {
        // UYV values from UYVY frame
        uint8_t U = input[(i << 2)];
        uint8_t Y1 = input[(i << 2) + 1];
        uint8_t V = input[(i << 2) + 2];
        uint8_t Y2 = input[(i << 2) + 3];

        uint8_t pixel1U = lut[256 * 256 * 3 * U + 256 * 3 * Y1 + 3 * V];
        uint8_t pixel1Y = lut[256 * 256 * 3 * U + 256 * 3 * Y1 + 3 * V + 1];
        uint8_t pixel1V = lut[256 * 256 * 3 * U + 256 * 3 * Y1 + 3 * V + 2];

        uint8_t pixel2U = lut[256 * 256 * 3 * U + 256 * 3 * Y2 + 3 * V];
        uint8_t pixel2Y = lut[256 * 256 * 3 * U + 256 * 3 * Y2 + 3 * V + 1];
        uint8_t pixel2V = lut[256 * 256 * 3 * U + 256 * 3 * Y2 + 3 * V + 2];

        // if (pixel1Y != pixel2Y) {
        //     printf("------------------------------------------------------------------------------------\n");
        //     printf("pixel1U: %d, pixel1V: %d, pixel2U: %d, pixel2V: %d\n", pixel1U, pixel1V, pixel2U, pixel2V);
        //     printf("------------------------------------------------------------------------------------\n");
        // }

        // getting corresponding LUT[U1][Y1][V1] values to put back into the frame
        output[(i << 2)] = (pixel1U + pixel2U) >> 1;
        output[(i << 2) + 1] = pixel1Y; 
        output[(i << 2) + 2] = (pixel1V + pixel2V) >> 1;
        output[(i << 2) + 3] = pixel2Y;

    }
}

// CUDA-accelerated function to apply LUT to the entire frame
uint8_t* applyLUTtoFrameCUDA(const uint8_t* frame, uint8_t* lut) {
    // Convert the frame to a vector of pixels
    int totalSize = H_BUFF * W_BUFF * 2;
    uint8_t* output = new uint8_t[totalSize];

    // Allocate GPU memory
    uint8_t* d_output;
    hipMalloc(&d_output, totalSize * sizeof(uint8_t));

    // Copy data to GPU
    hipMemcpy(d_output, output, totalSize * sizeof(uint8_t), hipMemcpyHostToDevice);

    // Launch the kernel to apply UYVY LUT to each pixel in UYVY frame
    applyLUTKernel<<<960, 256>>>(frame, d_output, W_BUFF * H_BUFF, lut);

    hipDeviceSynchronize();

    // check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "ERROR @ applying LUT: %s \n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Copy the result back to CPU
    hipMemcpy(output, d_output, totalSize * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_output);

    return output;
}