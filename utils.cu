#include "hip/hip_runtime.h"
#include "utils.h"

__global__ void cudauyvy2bgr(int framesize, uint8_t *input, uint8_t *output) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	framesize >>= 1;
	for(int i = index; i < framesize; i += stride) {
		float u = input[(i << 2)];
		float y = input[(i << 2) + 1];
		float v = input[(i << 2) + 2];
		y -= 16;
		u -= 128;
		v -= 128;
		float red = y * 1.164 + v * 1.596;
		float green = y * 1.164 - u * 0.392 - v * 0.813;
		float blue = y * 1.164 + u * 2.017;
		if(red > 255) red = 255;
		else if(red < 0) red = 0;
		if(green > 255) green = 255;
		else if(green < 0) green = 0;
		if(blue > 255) blue = 255;
		else if(blue < 0) blue = 0;
		output[i * 6] = blue;
		output[i * 6 + 1] = green;
		output[i * 6 + 2] = red;

		y = input[(i << 2) + 3];
		y -= 16;
		red = y * 1.164 + v * 1.596;
		green = y * 1.164 - u * 0.392 - v * 0.813;
		blue = y * 1.164 + u * 2.017;
		if(red > 255) red = 255;
		else if(red < 0) red = 0;
		if(green > 255) green = 255;
		else if(green < 0) green = 0;
		if(blue > 255) blue = 255;
		else if(blue < 0) blue = 0;
		output[i * 6 + 3] = blue;
		output[i * 6 + 4] = green;
		output[i * 6 + 5] = red;
	}
}

__global__ void cudargblut2yuv(uint8_t *input, uint8_t *output) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int idz = blockIdx.z * blockDim.z + threadIdx.z;
	if (idx < 256 && idy < 256 && idz < 256) {
		int index = idx * 256 * 256 * 3 + idy * 256 * 3 + idz * 3;
		float red = input[index];
		float green = input[index + 1];
		float blue = input[index + 2];
		float y = 16 + 0.256 * red + 0.504 * green + 0.0979 * blue;
		float u = 128 + 0.439 * red - 0.368 * green - 0.0714 * blue;
		float v = 128 - 0.148 * red  - 0.291 * green + 0.439 * blue;
		
		// clamping 0-255
		if(y > 255) y = 255;
		else if(y < 0) y = 0;
		if(u < 0) u = 0;
		else if(u > 255) u = 255;
		if(v < 0) v = 0;
		else if(v > 255) v = 255;
		
		output[index] = u;
		output[index + 1] = y;
		output[index + 2] = v;
	}
}
