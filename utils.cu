#include "hip/hip_runtime.h"
#include "utils.h"

__global__ void cudauyvy2bgr(int framesize, uint8_t *input, uint8_t *output) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	framesize >>= 1;
	for(int i = index; i < framesize; i += stride) {
		float u = input[(i << 2)];
		float y = input[(i << 2) + 1];
		float v = input[(i << 2) + 2];
		y -= 16;
		u -= 128;
		v -= 128;
		float red = y * 1.164 + v * 1.596;
		float green = y * 1.164 - u * 0.392 - v * 0.813;
		float blue = y * 1.164 + u * 2.017;
		if(red > 255) red = 255;
		else if(red < 0) red = 0;
		if(green > 255) green = 255;
		else if(green < 0) green = 0;
		if(blue > 255) blue = 255;
		else if(blue < 0) blue = 0;
		output[i * 6] = blue;
		output[i * 6 + 1] = green;
		output[i * 6 + 2] = red;

		y = input[(i << 2) + 3];
		y -= 16;
		red = y * 1.164 + v * 1.596;
		green = y * 1.164 - u * 0.392 - v * 0.813;
		blue = y * 1.164 + u * 2.017;
		if(red > 255) red = 255;
		else if(red < 0) red = 0;
		if(green > 255) green = 255;
		else if(green < 0) green = 0;
		if(blue > 255) blue = 255;
		else if(blue < 0) blue = 0;
		output[i * 6 + 3] = blue;
		output[i * 6 + 4] = green;
		output[i * 6 + 5] = red;
	}
}

__global__ void cudargblut2yuv(uint8_t *inputlut, uint8_t *outputlut, uint8_t* rgbIdx2yuvLut) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int idz = blockIdx.z * blockDim.z + threadIdx.z;
	if (idx < 256 && idy < 256 && idz < 256) {
		// index is the RGB value of a pixel we are trying to transform and resultant is the colour corrected RGB value
		int index = idx * 256 * 256 * 3 + idy * 256 * 3 + idz * 3;
		uint8_t red = inputlut[index + 0];
		uint8_t green = inputlut[index + 1];
		uint8_t blue = inputlut[index + 2];
		
		// RGB index for getting the corresponding YUV values for the resultant LUT RGB values
		int rgbIndex = 256 * 256 * 3 * red + 256 * 3 * green + 3 * blue;
		
		float y = rgbIdx2yuvLut[rgbIndex + 0];
		float u = rgbIdx2yuvLut[rgbIndex + 1];
		float v = rgbIdx2yuvLut[rgbIndex + 2];

		// now getting the appropriate YUV indexes for the colour corrected YUV values which we just fetched
		int yuvIdx = rgbIdx2yuvLut[index + 0] * 256 * 256 * 3 + rgbIdx2yuvLut[index + 1] * 256 * 3 + rgbIdx2yuvLut[index + 2] * 3;

		
		// use the cudayuvlut2rgb function to get the proper indexing
		outputlut[yuvIdx] = y;
		outputlut[yuvIdx + 1] = u;
		outputlut[yuvIdx + 2] = v;
	}
}

// Creating an RGB to YUV LUT with all possible values of RGB 0 to 255
__global__ void cudargbIdx2yuv(uint8_t* output) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int idz = blockIdx.z * blockDim.z + threadIdx.z;
	if (idx < 256 && idy < 256 && idz < 256) {

		int index = idx * 256 * 256 * 3 + idy * 256 * 3 + idz * 3;
		uint8_t red = idx;
		uint8_t green = idy;
		uint8_t blue = idz;

		// red, green, blue are the indexes which give the corresponding YUV values
		float y = 16 + 0.256 * red + 0.504 * green + 0.0979 * blue;
		float u = 128 + 0.439 * red - 0.368 * green - 0.0714 * blue;
		float v = 128 - 0.148 * red  - 0.291 * green + 0.439 * blue;
		
		// clamping 0-255
		if(y > 255) y = 255;
		else if(y < 0) y = 0;
		if(u < 0) u = 0;
		else if(u > 255) u = 255;
		if(v < 0) v = 0;
		else if(v > 255) v = 255;
		
		output[index] = y;
		output[index + 1] = u;
		output[index + 2] = v;
	}
}
